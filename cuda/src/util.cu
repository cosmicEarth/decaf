#include <hipblas.h>
#include "util.cuh"

int init_cuda() {
    hipError_t err;
    hipblasStatus_t cublas_err;
    err = hipDeviceSetCacheConfig(hipFuncCachePreferShared);
    CUDA_RETURN_ON_FAILURE(err);
    cublas_err = cublasInit();
    CUBLAS_RETURN_ON_FAILURE(cublas_err);
    return hipSuccess;
}
